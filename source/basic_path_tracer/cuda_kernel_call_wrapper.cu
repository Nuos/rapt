/* rapt - RichieSam's Adventures in Path Tracing
 *
 * rapt is the legal property of Adrian Astley
 * Copyright Adrian Astley 2015
 */

#include "basic_path_tracer/path_trace.cuh"


uint32 WangHash(uint32 a) {
    a = (a ^ 61) ^ (a >> 16);
    a = a + (a << 3);
    a = a ^ (a >> 4);
    a = a * 0x27d4eb2d;
    a = a ^ (a >> 15);
    return a;
}

void PathTraceNextFrame(void *buffer, uint width, uint height, size_t pitch, DeviceCamera *camera, Scene::Sphere *spheres, uint numSpheres, Scene::LambertMaterial *g_materials, uint numMaterials, uint frameNumber) {
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);

	PathTraceKernel<<<Dg, Db>>>((unsigned char *)buffer, width, height, pitch, camera, spheres, numSpheres, g_materials, numMaterials, WangHash(frameNumber));

	error = hipGetLastError();
	if (error != hipSuccess) {
		//exit(error);
	}
}
