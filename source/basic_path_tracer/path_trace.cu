#include "hip/hip_runtime.h"
/* rapt - RichieSam's Adventures in Path Tracing
 *
 * rapt is the legal property of Adrian Astley
 * Copyright Adrian Astley 2015
 */

#include "basic_path_tracer/path_trace.cuh"

#include "scene/scene_objects.h"

#include <>
#include <graphics/hip/hip_vector_types.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_vector_types.h>

#include <float.h>


__device__ float3 CalculateRayDirectionFromPixel(uint x, uint y, uint width, uint height, DeviceCamera &camera, hiprandState *randState) {
	float3 viewVector = make_float3((((x + hiprand_uniform(randState)) / width) * 2.0f - 1.0f) * camera.TanFovDiv2_X,
	                                -(((y + hiprand_uniform(randState)) / height) * 2.0f - 1.0f) * camera.TanFovDiv2_Y,
	                                1.0f);

	// Matrix multiply
	return normalize(make_float3(dot(viewVector, camera.ViewToWorldMatrixR0),
	                             dot(viewVector, camera.ViewToWorldMatrixR1),
	                             dot(viewVector, camera.ViewToWorldMatrixR2)));
}

/**
 * Test for the intersection of a ray with a sphere
 *
 * NOTE: Source adapted from Scratchapixel.com Lesson 7 - Intersecting Simple Shapes
 *       http://www.scratchapixel.com/old/lessons/3d-basic-lessons/lesson-7-intersecting-simple-shapes/ray-sphere-intersection/
 *
 * @param ray       The ray
 * @param sphere    The sphere
 * @return          The distance from the ray origin to the nearest intersection. -1.0f if no intersection
 */
__device__ float TestRaySphereIntersection(Scene::Ray &ray, Scene::Sphere &sphere, float3 &normal_out) {
	float3 L = sphere.Center - ray.Origin;
    float projectedRay = dot(L, ray.Direction);

	// Ray points away from the sphere
	if (projectedRay < 0) {
		return -1.0f;
	}

    float distanceToRaySquared = dot(L, L) - projectedRay * projectedRay;

	// Ray misses the sphere
    if (distanceToRaySquared > sphere.RadiusSquared) {
		return -1.0f;
	}

	// See http://www.scratchapixel.com/old/assets/Uploads/Lesson007/l007-raysphereisect1.png for definition of thc
    float thc = sqrt(sphere.RadiusSquared - distanceToRaySquared);

    float firstIntersection = projectedRay - thc;
    float secondIntersection = projectedRay + thc;

	float nearestIntersection;
	float normalDirection;
	if (firstIntersection > 0 && secondIntersection > 0) {
		// Two intersections
		// Return the nearest of the two
		nearestIntersection = min(firstIntersection, secondIntersection);

		normalDirection = 1.0f;
	} else {
		// Ray starts inside the sphere
		// Return the far side of the sphere
		nearestIntersection = max(firstIntersection, secondIntersection);

		// We reverse the direction of the normal, since we are inside the sphere
		normalDirection = -1.0f;
	}

	normal_out = normalize(((ray.Origin + (ray.Direction * nearestIntersection)) - sphere.Center) * normalDirection);

	return nearestIntersection;
}

__global__ void PathTraceKernel(unsigned char *textureData, uint width, uint height, size_t pitch, DeviceCamera *g_camera, Scene::Sphere *g_spheres, uint numSpheres, uint hashedFrameNumber) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	// Create a local copy of the camera
	DeviceCamera camera = *g_camera;

	// Global threadId
	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	// Create random number generator
	hiprandState randState;
	hiprand_init(hashedFrameNumber + threadId, 0, 0, &randState);

	// Calculate the first ray for this pixel
	Scene::Ray ray = {camera.Origin, CalculateRayDirectionFromPixel(x, y, width, height, camera, &randState)};

	// Generate a uniform random number
	//float randNum = hiprand_uniform(&randState);

	// Try to intersect with the spheres;
	float closestIntersection = FLT_MAX;
	float3 normal;
	for (uint i = 0; i < numSpheres; ++i) {
		float3 newNormal;
		float intersection = TestRaySphereIntersection(ray, g_spheres[i], newNormal);
		if (intersection > 0.0f && intersection < closestIntersection) {
			closestIntersection = intersection;
			normal = newNormal;
		}
	}

	float3 pixelColor;
	if (closestIntersection < FLT_MAX) {
		float attentuation = max(dot(normal, make_float3(0.70710678118f, 0.70710678118f, -0.70710678118f)), 0.0f);
		pixelColor = make_float3(0.846, 0.933, 0.949) * attentuation + make_float3(0.15f, 0.15f, 0.15f);
	} else {
		pixelColor = make_float3(0.0f, 0.0f, 0.0f);
	}

	if (x < width && y < height) {
		// Get a pointer to the pixel at (x,y)
		float *pixel = (float *)(textureData + y * pitch) + 4 /*RGBA*/ * x;

		// Write out pixel data
		pixel[0] += pixelColor.x;
		pixel[1] += pixelColor.y;
		pixel[2] += pixelColor.z;
		// Ignore alpha, since it's hardcoded to 1.0f in the display
		// We have to use a RGBA format since CUDA-DirectX interop doesn't support R32G32B32_FLOAT
	}
}
