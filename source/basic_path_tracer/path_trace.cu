#include "hip/hip_runtime.h"
/* rapt - RichieSam's Adventures in Path Tracing
 *
 * rapt is the legal property of Adrian Astley
 * Copyright Adrian Astley 2015
 */

#include "basic_path_tracer/path_trace.cuh"

#include "scene/scene_objects.h"

#include <>
#include <graphics/hip/hip_vector_types.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_vector_types.h>

#include <float.h>


__device__ float3 CalculateRayDirectionFromPixel(uint x, uint y, uint width, uint height, DeviceCamera &camera) {
	float3 viewVector = make_float3((((x + 0.5f /*TODO: Add jitter */) / width) * 2.0f - 1.0f) * camera.TanFovDiv2_X,
	                                -(((y + 0.5f /*TODO: Add jitter */) / height) * 2.0f - 1.0f) * camera.TanFovDiv2_Y,
	                                1.0f);

	// Matrix multiply
	return normalize(make_float3(dot(viewVector, camera.X),
	                             dot(viewVector, camera.Y),
	                             dot(viewVector, camera.Z)));
}

/**
 * Test for the intersection of a ray with a sphere
 *
 * NOTE: Source adapted from Scratchapixel.com Lesson 7 - Intersecting Simple Shapes
 *       http://www.scratchapixel.com/old/lessons/3d-basic-lessons/lesson-7-intersecting-simple-shapes/ray-sphere-intersection/
 *
 * @param ray       The ray
 * @param sphere    The sphere
 * @return          The distance from the ray origin to the nearest intersection. -1.0f if no intersection
 */
__device__ float TestRaySphereIntersection(Scene::Ray &ray, Scene::Sphere &sphere) {
	float3 L = sphere.Center - ray.Origin;
    float projectedRay = dot(L, ray.Direction);

	// Ray points away from the sphere
	if (projectedRay < 0) {
		return -1.0f;
	}

    float distanceToRaySquared = dot(L, L) - projectedRay * projectedRay;

	// Ray misses the sphere
    if (distanceToRaySquared > sphere.RadiusSquared) {
		return -1.0f;
	}

	// See http://www.scratchapixel.com/old/assets/Uploads/Lesson007/l007-raysphereisect1.png for definition of thc
    float thc = sqrt(sphere.RadiusSquared - distanceToRaySquared);

    float firstIntersection = projectedRay - thc;
    float secondIntersection = projectedRay + thc;

	float nearestIntersection;
	if (firstIntersection > 0 && secondIntersection > 0) {
		// Two intersections
		// Return the nearest of the two
		nearestIntersection = min(firstIntersection, secondIntersection);
	} else {
		// Ray starts inside the sphere
		// Return the far side of the sphere
		nearestIntersection = max(firstIntersection, secondIntersection);
	}

	return nearestIntersection;
}

__global__ void PathTraceKernel(unsigned char *textureData, uint width, uint height, size_t pitch, DeviceCamera camera, Scene::Sphere *spheres, uint numSpheres, uint hashedFrameNumber) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= width || y >= height) {
		return;
	}

	// Global threadId
	int threadId = (blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

	// Create random number generator
	//hiprandState randState;
	//hiprand_init(hashedFrameNumber + threadId, 0, 0, &randState);

	// Calculate the first ray for this pixel
	Scene::Ray ray = {camera.Origin, CalculateRayDirectionFromPixel(x, y, width, height, camera)};

	// Generate a uniform random number
	//float randNum = hiprand_uniform(&randState);

	// Try to intersect with the spheres;
	float closestIntersection = FLT_MAX;
	for (uint i = 0; i < numSpheres; ++i) {
		float intersection = TestRaySphereIntersection(ray, spheres[i]);
		if (intersection > 0.0f) {
			closestIntersection = min(closestIntersection, intersection);
		}
	}

	float pixelColor;
	if (closestIntersection == FLT_MAX) {
		pixelColor = 0.0f;
	} else {
		pixelColor = 1.0f - (closestIntersection * 0.05f);
	}

	if (x < width && y < height) {
		// Get a pointer to the pixel at (x,y)
		float *pixel = (float *)(textureData + y * pitch) + 4 /*RGBA*/ * x;

		// Write out pixel data
			pixel[0] += pixelColor;
			pixel[1] += pixelColor;
			pixel[2] += pixelColor;
			// Ignore alpha, since it's hardcoded to 1.0f in the display
			// We have to use a RGBA format since CUDA-DirectX interop doesn't support R32G32B32_FLOAT
	}
}
